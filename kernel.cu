#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <omp.h>

#define Zad1


#ifdef Zad1
int coresPerSM(hipDeviceProp_t prop) {
	typedef struct {
		int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
		// and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
		{0x30, 192},
		{0x32, 192},
		{0x35, 192},
		{0x37, 192},
		{0x50, 128},
		{0x52, 128},
		{0x53, 128},
		{0x60,  64},
		{0x61, 128},
		{0x62, 128},
		{0x70,  64},
		{0x72,  64},
		{0x75,  64},
		{-1, -1} };

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((prop.major << 4) + prop.minor)) {
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one
	// to run properly
	printf(
		"MapSMtoCores for SM %d.%d is undefined."
		"  Default to use %d Cores/SM\n",
		prop.major, prop.minor, nGpuArchCoresPerSM[index - 1].Cores);
	return nGpuArchCoresPerSM[index - 1].Cores;
}

int main()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("Nazwa urzadzenia: %s\n", prop.name);
	printf("Czestotliwosc zegara [KHz]: %d\n", prop.memoryClockRate);
	printf("Przepustowosc pamieci [bity]: %d\n", prop.memoryBusWidth);
	printf("Compute Capability: %d\n", coresPerSM(prop));
	printf("Liczba multiprocesorow: %d\n", prop.multiProcessorCount);
	printf("Liczba rdzeni: %d\n", (coresPerSM(prop)) * prop.multiProcessorCount);

	hipSetDevice(0);
	hipDeviceReset();
    return 0;
}
#endif

#ifdef Zad2

int main()
{
	hipSetDevice(0);

	char *charmib1, *charmib8, *charmib96, *charmib256, *a1, *a8, *a96, *a256;
	int *intmib1, *intmib8, *intmib96, *intmib256, *b1, *b8, *b96, *b256;
	float *floatmib1, *floatmib8, *floatmib96, *floatmib256, *c1, *c8, *c96, *c256;
	double *doublemib1, *doublemib8, *doublemib96, *doublemib256, *d1, *d8, *d96, *d256;
	charmib1 = new char[1024 * 1024];
	charmib8 = new char[8 * 1024 * 1024];
	charmib96 = new char[96 * 1024 * 1024];
	charmib256 = new char[256 * 1024 * 1024];
	intmib1 = new int[1024 * 1024 / 4];
	intmib8 = new int[2 * 1024 * 1024];
	intmib96 = new int[24 * 1024 * 1024];
	intmib256 = new int[64 * 1024 * 1024];
	floatmib1 = new float[1024 * 1024 / 4];
	floatmib8 = new float[2 * 1024 * 1024];
	floatmib96 = new float[24 * 1024 * 1024];
	floatmib256 = new float[64 * 1024 * 1024];
	doublemib1 = new double[1024 * 1024 / 8];
	doublemib8 = new double[1024 * 1024];
	doublemib96 = new double[12 * 1024 * 1024];
	doublemib256 = new double[32 * 1024 * 1024];

	hipMalloc(&a1, 1024 * 1024 * sizeof(char));
	hipMalloc(&a8, 1024 * 1024 * 8 * sizeof(char));
	hipMalloc(&a96, 1024 * 1024 * 96 * sizeof(char));
	hipMalloc(&a256, 1024 * 1024 * 256 * sizeof(char));
	hipMalloc(&b1, 1024 * 1024 * sizeof(int) / 4);
	hipMalloc(&b8, 1024 * 1024 * 2 * sizeof(int));
	hipMalloc(&b96, 1024 * 1024 * 24 * sizeof(int));
	hipMalloc(&b256, 1024 * 1024 * 64 * sizeof(int));
	hipMalloc(&c1, 1024 * 1024 * sizeof(float) / 4);
	hipMalloc(&c8, 1024 * 1024 * 2 * sizeof(float));
	hipMalloc(&c96, 1024 * 1024 * 24 * sizeof(float));
	hipMalloc(&c256, 1024 * 1024 * 64 * sizeof(float));
	hipMalloc(&d1, 1024 * 1024 * sizeof(double) / 8);
	hipMalloc(&d8, 1024 * 1024 * sizeof(double));
	hipMalloc(&d96, 1024 * 1024 * 12 * sizeof(double));
	hipMalloc(&d256, 1024 * 1024 * 32 * sizeof(double));
	
	hipEvent_t start, stop;
	float czas;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipMemcpy(a1, charmib1, 1024 * 1024 * sizeof(char), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (char, 1 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(a8, charmib8, 1024 * 1024 * 8 *sizeof(char), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (char, 8 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(a96, charmib96, 1024 * 1024 * 96 *sizeof(char), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (char, 96 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(a256, charmib256, 1024 * 1024 * 256 * sizeof(char), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (char, 256 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(b1, intmib1, 1024 * 1024 * sizeof(int) / 4, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (int, 1 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(b8, intmib8, 1024 * 1024 * 2 * sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (int, 8 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(b96, intmib96, 1024 * 1024 * 24 * sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (int, 96 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(b256, intmib256, 1024 * 1024 * 64 * sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (int, 256 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(c1, floatmib1, 1024 * 1024 * sizeof(float) / 4, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (float, 1 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(c8, floatmib8, 1024 * 1024 * 2 * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (float, 8 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(c96, floatmib96, 1024 * 1024 * 24 * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (float, 96 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(c256, floatmib256, 1024 * 1024 * 64 * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (float, 256 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(d1, doublemib1, 1024 * 1024 * sizeof(double) / 8, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (double, 1 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(d8, doublemib8, 1024 * 1024 * sizeof(double), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (double, 8 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(d96, doublemib96, 1024 * 1024 * 12 * sizeof(double), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (double, 96 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(d256, doublemib256, 1024 * 1024 * 32 * sizeof(double), hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania HostToDevice (double, 256 MiB) [ms]: %f\n\n", czas);

	hipEventRecord(start, 0);
	hipMemcpy(charmib1, a1, 1024 * 1024 * sizeof(char), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (char, 1 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(charmib8, a8, 1024 * 1024 * 8 * sizeof(char), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (char, 8 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(charmib96, a96, 1024 * 1024 * 64 * sizeof(char), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (char, 96 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(charmib256, a256, 1024 * 1024 * 256 * sizeof(char), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (char, 256 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(intmib1, b1, 1024 * 1024 * sizeof(int) / 4, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (int, 1 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(intmib8, b8, 1024 * 1024 * 2 * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (int, 8 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(intmib96, b96, 1024 * 1024 * 24 * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (int, 96 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(intmib256, b256, 1024 * 1024 * 64 * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (int, 256 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(floatmib1, c1, 1024 * 1024 * sizeof(float) / 4, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (float, 1 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(floatmib8, c8, 1024 * 1024 * 2 * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (float, 8 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(floatmib96, c96, 1024 * 1024 * 24 * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (float, 96 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(floatmib256, c256, 1024 * 1024 * 64 * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (float, 256 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(doublemib1, d1, 1024 * 1024 * sizeof(double) / 8, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (double, 1 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(doublemib8, d8, 1024 * 1024 * sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (double, 8 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(doublemib96, d96, 1024 * 1024 * 12 * sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (double, 96 MiB) [ms]: %f\n", czas);
	hipEventRecord(start, 0);
	hipMemcpy(doublemib256, d256, 1024 * 1024 * 32 * sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&czas, start, stop);
	printf("Czas przesylania DeviceToHost (double, 256 MiB) [ms]: %f\n", czas);

	delete[] charmib1;
	delete[] charmib8;
	delete[] charmib96;
	delete[] charmib256;
	delete[] intmib1;
	delete[] intmib8;
	delete[] intmib96;
	delete[] intmib256;
	delete[] floatmib1;
	delete[] floatmib8;
	delete[] floatmib96;
	delete[] floatmib256;
	delete[] doublemib1;
	delete[] doublemib8;
	delete[] doublemib96;
	delete[] doublemib256;

	hipFree(a1);
	hipFree(a8);
	hipFree(a96);
	hipFree(a256);
	hipFree(b1);
	hipFree(b8);
	hipFree(b96);
	hipFree(b256);
	hipFree(c1);
	hipFree(c8);
	hipFree(c96);
	hipFree(c256);
	hipFree(d1);
	hipFree(d8);
	hipFree(d96);
	hipFree(d256);
	hipDeviceReset();
	return 0;
}
#endif

#ifdef Zad3

__global__ void kernelMnozenie(int *a, int *b, int *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	c[i] = a[i] * b[i];
}

__global__ void kernelDodawanie(int *a, int *b, int *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void kernelPotegowanie(int *a, int *b, int *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int wynik = 1;
	for (int j = 0; j < b[i]; j++)
	{
		wynik *= a[i];
	}
	c[i] = wynik;
}

__global__ void kernelMnozenie(float *a, float *b, float *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	c[i] = a[i] * b[i];
}

__global__ void kernelDodawanie(float *a, float *b, float *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void kernelPotegowanie(float *a, float *b, float *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	float wynik = 1;
	for (float j = 0; j < b[i]; j++)
	{
		wynik *= a[i];
	}
	c[i] = wynik;
}

__global__ void kernelMnozenie(double *a, double *b, double *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	c[i] = a[i] * b[i];
}

__global__ void kernelDodawanie(double *a, double *b, double *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void kernelPotegowanie(double *a, double *b, double *c)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	double wynik = 1;
	for (double j = 0; j < b[i]; j++)
	{
		wynik *= a[i];
	}
	c[i] = wynik;
}

void dodawanieCPU(int *a, int *b, int *c, int rozmiar);
void mnozenieCPU(int *a, int *b, int *c, int rozmiar);
void potegowanieCPU(int *a, int *b, int *c, int rozmiar);
void dodawanieCPU(float *a, float *b, float *c, int rozmiar);
void mnozenieCPU(float *a, float *b, float *c, int rozmiar);
void potegowanieCPU(float *a, float *b, float *c, int rozmiar);
void dodawanieCPU(double *a, double *b, double *c, int rozmiar);
void mnozenieCPU(double *a, double *b, double *c, int rozmiar);
void potegowanieCPU(double *a, double *b, double *c, int rozmiar);

int main()
{
	hipSetDevice(0);
	
	/*const int rozmiar = 9;
	int a[rozmiar] = { 1, 2, 3, 4, 5, 6, 7, 8, 9 };
	int b[rozmiar] = { 1, 2, 3, 4, 5, 6, 7, 8, 9 };
	int c[rozmiar] = { 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	int *dev_a;
	int *dev_b;
	int *dev_c;
	
	hipMalloc(&dev_a, rozmiar * sizeof(int));
	hipMalloc(&dev_b, rozmiar * sizeof(int));
	hipMalloc(&dev_c, rozmiar * sizeof(int));
	hipMemcpy(dev_a, a, rozmiar * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, rozmiar * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, rozmiar * sizeof(int), hipMemcpyHostToDevice);
	kernelDodawanie << <1, rozmiar >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, rozmiar * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nDodawanie GPU\n");
	for (int i = 0; i < rozmiar; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	kernelMnozenie << <1, rozmiar >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, rozmiar * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nMnozenie GPU\n");
	for (int i = 0; i < rozmiar; i++)
	{
		printf("%d * %d = %d\n", a[i], b[i], c[i]);
	}
	kernelPotegowanie << <1, rozmiar >> > (dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, rozmiar * sizeof(int), hipMemcpyDeviceToHost);
	printf("\nPotegowanie GPU\n");
	for (int i = 0; i < rozmiar; i++)
	{
		printf("%d ^ %d = %d\n", a[i], b[i], c[i]);
	}
	dodawanieCPU(a, b, c, rozmiar);
	printf("\nDodawanie CPU\n");
	for (int i = 0; i < rozmiar; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	mnozenieCPU(a, b, c, rozmiar);
	printf("\nMnozenie CPU\n");
	for (int i = 0; i < rozmiar; i++)
	{
		printf("%d * %d = %d\n", a[i], b[i], c[i]);
	}
	potegowanieCPU(a, b, c, rozmiar);
	printf("\nPotegowanie CPU\n");
	for (int i = 0; i < rozmiar; i++)
	{
		printf("%d ^ %d = %d\n", a[i], b[i], c[i]);
	}
	*/

	const int rozmiar = 1024 * 1024;
	int liczbaBlokow;
	int rozmiarBloku = 1024;
	int *aint1 = new int[rozmiar / 4];
	int *bint1 = new int[rozmiar / 4];
	int *cint1 = new int[rozmiar / 4];
	int *aint4 = new int[rozmiar];
	int *bint4 = new int[rozmiar];
	int *cint4 = new int[rozmiar];
	int *aint8 = new int[rozmiar * 2];
	int *bint8 = new int[rozmiar * 2];
	int *cint8 = new int[rozmiar * 2];
	int *aint16 = new int[rozmiar * 4];
	int *bint16 = new int[rozmiar * 4];
	int *cint16 = new int[rozmiar * 4];
	float *afloat1 = new float[rozmiar / 4];
	float *bfloat1 = new float[rozmiar / 4];
	float *cfloat1 = new float[rozmiar / 4];
	float *afloat4 = new float[rozmiar];
	float *bfloat4 = new float[rozmiar];
	float *cfloat4 = new float[rozmiar];
	float *afloat8 = new float[rozmiar * 2];
	float *bfloat8 = new float[rozmiar * 2];
	float *cfloat8 = new float[rozmiar * 2];
	float *afloat16 = new float[rozmiar * 4];
	float *bfloat16 = new float[rozmiar * 4];
	float *cfloat16 = new float[rozmiar * 4];
	double *adouble1 = new double[rozmiar / 8];
	double *bdouble1 = new double[rozmiar / 8];
	double *cdouble1 = new double[rozmiar / 8];
	double *adouble4 = new double[rozmiar / 2];
	double *bdouble4 = new double[rozmiar / 2];
	double *cdouble4 = new double[rozmiar / 2];
	double *adouble8 = new double[rozmiar];
	double *bdouble8 = new double[rozmiar];
	double *cdouble8 = new double[rozmiar];
	double *adouble16 = new double[rozmiar * 2];
	double *bdouble16 = new double[rozmiar * 2];
	double *cdouble16 = new double[rozmiar * 2];
	int *dev_aint1;
	int *dev_aint4;
	int *dev_aint8;
	int *dev_aint16;
	int *dev_bint1;
	int *dev_bint4;
	int *dev_bint8;
	int *dev_bint16;
	int *dev_cint1;
	int *dev_cint4;
	int *dev_cint8;
	int *dev_cint16;
	float *dev_afloat1;
	float *dev_afloat4;
	float *dev_afloat8;
	float *dev_afloat16;
	float *dev_bfloat1;
	float *dev_bfloat4;
	float *dev_bfloat8;
	float *dev_bfloat16;
	float *dev_cfloat1;
	float *dev_cfloat4;
	float *dev_cfloat8;
	float *dev_cfloat16;
	double *dev_adouble1;
	double *dev_adouble4;
	double *dev_adouble8;
	double *dev_adouble16;
	double *dev_bdouble1;
	double *dev_bdouble4;
	double *dev_bdouble8;
	double *dev_bdouble16;
	double *dev_cdouble1;
	double *dev_cdouble4;
	double *dev_cdouble8;
	double *dev_cdouble16;
	
	hipMalloc(&dev_aint1, rozmiar * sizeof(int) / 4);
	hipMalloc(&dev_aint4, rozmiar * sizeof(int));
	hipMalloc(&dev_aint8, rozmiar * sizeof(int) * 2);
	hipMalloc(&dev_aint16, rozmiar * sizeof(int) * 4);
	hipMalloc(&dev_bint1, rozmiar * sizeof(int) / 4);
	hipMalloc(&dev_bint4, rozmiar * sizeof(int));
	hipMalloc(&dev_bint8, rozmiar * sizeof(int) * 2);
	hipMalloc(&dev_bint16, rozmiar * sizeof(int) * 4);
	hipMalloc(&dev_cint1, rozmiar * sizeof(int)) / 4;
	hipMalloc(&dev_cint4, rozmiar * sizeof(int));
	hipMalloc(&dev_cint8, rozmiar * sizeof(int) * 2);
	hipMalloc(&dev_cint16, rozmiar * sizeof(int) * 4);
	hipMalloc(&dev_afloat1, rozmiar * sizeof(float) / 4);
	hipMalloc(&dev_afloat4, rozmiar * sizeof(float));
	hipMalloc(&dev_afloat8, rozmiar * sizeof(float) * 2);
	hipMalloc(&dev_afloat16, rozmiar * sizeof(float) * 4);
	hipMalloc(&dev_bfloat1, rozmiar * sizeof(float) / 4);
	hipMalloc(&dev_bfloat4, rozmiar * sizeof(float));
	hipMalloc(&dev_bfloat8, rozmiar * sizeof(float) * 2);
	hipMalloc(&dev_bfloat16, rozmiar * sizeof(float) * 4);
	hipMalloc(&dev_cfloat1, rozmiar * sizeof(float) / 4);
	hipMalloc(&dev_cfloat4, rozmiar * sizeof(float));
	hipMalloc(&dev_cfloat8, rozmiar * sizeof(float) * 2);
	hipMalloc(&dev_cfloat16, rozmiar * sizeof(float) * 4);
	hipMalloc(&dev_adouble1, rozmiar * sizeof(double) / 8);
	hipMalloc(&dev_adouble4, rozmiar * sizeof(double) / 2);
	hipMalloc(&dev_adouble8, rozmiar * sizeof(double));
	hipMalloc(&dev_adouble16, rozmiar * sizeof(double) * 2);
	hipMalloc(&dev_bdouble1, rozmiar * sizeof(double) / 8);
	hipMalloc(&dev_bdouble4, rozmiar * sizeof(double) / 2);
	hipMalloc(&dev_bdouble8, rozmiar * sizeof(double));
	hipMalloc(&dev_bdouble16, rozmiar * sizeof(double) * 2);
	hipMalloc(&dev_cdouble1, rozmiar * sizeof(double) / 8);
	hipMalloc(&dev_cdouble4, rozmiar * sizeof(double) / 2);
	hipMalloc(&dev_cdouble8, rozmiar * sizeof(double));
	hipMalloc(&dev_cdouble16, rozmiar * sizeof(double) * 2);

	hipMemcpy(dev_aint1, aint1, rozmiar * sizeof(int) / 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_aint4, aint4, rozmiar * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_aint8, aint8, rozmiar * sizeof(int) * 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_aint16, aint16, rozmiar * sizeof(int) * 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_bint1, bint1, rozmiar * sizeof(int) / 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_bint4, bint4, rozmiar * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_bint8, bint8, rozmiar * sizeof(int) * 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_bint16, bint16, rozmiar * sizeof(int) * 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_cint1, cint1, rozmiar * sizeof(int) / 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_cint4, cint4, rozmiar * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_cint8, cint8, rozmiar * sizeof(int) * 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_cint16, cint16, rozmiar * sizeof(int) * 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_afloat1, afloat1, rozmiar * sizeof(float) / 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_afloat4, afloat4, rozmiar * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_afloat8, afloat8, rozmiar * sizeof(float) * 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_afloat16, afloat16, rozmiar * sizeof(float) * 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_bfloat1, bfloat1, rozmiar * sizeof(float) / 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_bfloat4, bfloat4, rozmiar * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_bfloat8, bfloat8, rozmiar * sizeof(float) * 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_bfloat16, bfloat16, rozmiar * sizeof(float) * 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_cfloat1, cfloat1, rozmiar * sizeof(float) / 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_cfloat4, cfloat4, rozmiar * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_cfloat8, cfloat8, rozmiar * sizeof(float) * 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_cfloat16, cfloat16, rozmiar * sizeof(float) * 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_adouble1, adouble1, rozmiar * sizeof(double) / 8, hipMemcpyHostToDevice);
	hipMemcpy(dev_adouble4, adouble4, rozmiar * sizeof(double) / 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_adouble8, adouble8, rozmiar * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_adouble16, adouble16, rozmiar * sizeof(double) * 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_bdouble1, bdouble1, rozmiar * sizeof(double) / 8, hipMemcpyHostToDevice);
	hipMemcpy(dev_bdouble4, bdouble4, rozmiar * sizeof(double) / 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_bdouble8, bdouble8, rozmiar * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_bdouble16, bdouble16, rozmiar * sizeof(double) * 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_cdouble1, cdouble1, rozmiar * sizeof(double) / 8, hipMemcpyHostToDevice);
	hipMemcpy(dev_cdouble4, cdouble4, rozmiar * sizeof(double) / 2, hipMemcpyHostToDevice);
	hipMemcpy(dev_cdouble8, cdouble8, rozmiar * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_cdouble16, cdouble16, rozmiar * sizeof(double) * 2, hipMemcpyHostToDevice);

	float czasGPU;
	hipEvent_t startGPU, stopGPU;
	double startCPU, stopCPU;

	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_aint1, dev_bint1, dev_cint1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint1, dev_cint1, rozmiar * sizeof(int) / 4, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (int, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_aint4, dev_bint4, dev_cint4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint4, dev_cint4, rozmiar * sizeof(int), hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (int, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_aint8, dev_bint8, dev_cint8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint8, dev_cint8, rozmiar * sizeof(int) * 2, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (int, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_aint16, dev_bint16, dev_cint16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint16, dev_cint16, rozmiar * sizeof(int) * 4, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (int, 16MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat1, dev_bfloat1, dev_cfloat1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat1, dev_cfloat1, rozmiar * sizeof(float) / 4, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (float, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat4, dev_bfloat4, dev_cfloat4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat4, dev_cfloat4, rozmiar * sizeof(float), hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (float, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat8, dev_bfloat8, dev_cfloat8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat8, dev_cfloat8, rozmiar * sizeof(float) * 2, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (float, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat16, dev_bfloat16, dev_cfloat16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat16, dev_cfloat16, rozmiar * sizeof(float) * 4, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (float, 16MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 8 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble1, dev_bdouble1, dev_cdouble1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble1, dev_cdouble1, rozmiar * sizeof(double) / 8, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (double, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble4, dev_bdouble4, dev_cdouble4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble4, dev_cdouble4, rozmiar * sizeof(double) / 2, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (double, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble8, dev_bdouble8, dev_cdouble8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble8, dev_cdouble8, rozmiar * sizeof(double), hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (double, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelDodawanie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble16, dev_bdouble16, dev_cdouble16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble16, dev_cdouble16, rozmiar * sizeof(double) * 2, hipMemcpyDeviceToHost);
	printf("Czas dodawania GPU (double, 16MiB) [ms]: %f\n\n", czasGPU);

	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_aint1, dev_bint1, dev_cint1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint1, dev_cint1, rozmiar * sizeof(int) / 4, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (int, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_aint4, dev_bint4, dev_cint4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint4, dev_cint4, rozmiar * sizeof(int), hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (int, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_aint8, dev_bint8, dev_cint8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint8, dev_cint8, rozmiar * sizeof(int) * 2, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (int, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_aint16, dev_bint16, dev_cint16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint16, dev_cint16, rozmiar * sizeof(int) * 4, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (int, 16MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat1, dev_bfloat1, dev_cfloat1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat1, dev_cfloat1, rozmiar * sizeof(float) / 4, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (float, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat4, dev_bfloat4, dev_cfloat4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat4, dev_cfloat4, rozmiar * sizeof(float), hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (float, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat8, dev_bfloat8, dev_cfloat8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat8, dev_cfloat8, rozmiar * sizeof(float) * 2, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (float, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat16, dev_bfloat16, dev_cfloat16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat16, dev_cfloat16, rozmiar * sizeof(float) * 4, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (float, 16MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 8 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble1, dev_bdouble1, dev_cdouble1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble1, dev_cdouble1, rozmiar * sizeof(double) / 8, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (double, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble4, dev_bdouble4, dev_cdouble4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble4, dev_cdouble4, rozmiar * sizeof(double) / 2, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (double, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble8, dev_bdouble8, dev_cdouble8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble8, dev_cdouble8, rozmiar * sizeof(double), hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (double, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelMnozenie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble16, dev_bdouble16, dev_cdouble16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble16, dev_cdouble16, rozmiar * sizeof(double) * 2, hipMemcpyDeviceToHost);
	printf("Czas mnozenia GPU (double, 16MiB) [ms]: %f\n\n", czasGPU);

	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_aint1, dev_bint1, dev_cint1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint1, dev_cint1, rozmiar * sizeof(int) / 4, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (int, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_aint4, dev_bint4, dev_cint4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint4, dev_cint4, rozmiar * sizeof(int), hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (int, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_aint8, dev_bint8, dev_cint8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint8, dev_cint8, rozmiar * sizeof(int) * 2, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (int, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_aint16, dev_bint16, dev_cint16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cint16, dev_cint16, rozmiar * sizeof(int) * 4, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (int, 16MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat1, dev_bfloat1, dev_cfloat1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat1, dev_cfloat1, rozmiar * sizeof(float) / 4, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (float, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat4, dev_bfloat4, dev_cfloat4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat4, dev_cfloat4, rozmiar * sizeof(float), hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (float, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat8, dev_bfloat8, dev_cfloat8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat8, dev_cfloat8, rozmiar * sizeof(float) * 2, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (float, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 4 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_afloat16, dev_bfloat16, dev_cfloat16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cfloat16, dev_cfloat16, rozmiar * sizeof(float) * 4, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (float, 16MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 8 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble1, dev_bdouble1, dev_cdouble1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble1, dev_cdouble1, rozmiar * sizeof(double) / 8, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (double, 1MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar / 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble4, dev_bdouble4, dev_cdouble4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble4, dev_cdouble4, rozmiar * sizeof(double) / 2, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (double, 4MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble8, dev_bdouble8, dev_cdouble8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble8, dev_cdouble8, rozmiar * sizeof(double), hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (double, 8MiB) [ms]: %f\n", czasGPU);
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	liczbaBlokow = (rozmiar * 2 - rozmiarBloku + 1) / rozmiarBloku;
	hipEventRecord(startGPU, 0);
	kernelPotegowanie << <liczbaBlokow, rozmiarBloku >> > (dev_adouble16, dev_bdouble16, dev_cdouble16);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	hipMemcpy(cdouble16, dev_cdouble16, rozmiar * sizeof(double) * 2, hipMemcpyDeviceToHost);
	printf("Czas potegowanie GPU (double, 16MiB) [ms]: %f\n\n", czasGPU);
 
	startCPU = omp_get_wtime();
	dodawanieCPU(aint1, bint1, cint1, rozmiar / 4);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (int, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(aint4, bint4, cint4, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (int, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(aint8, bint8, cint8, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (int, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(aint16, bint16, cint16, rozmiar * 4);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (int, 16MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(afloat1, bfloat1, cfloat1, rozmiar / 4);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (float, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(afloat4, bfloat4, cfloat4, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (float, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(afloat8, bfloat8, cfloat8, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (float, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(afloat16, bfloat16, cfloat16, rozmiar * 4);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (float, 16MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(adouble1, bdouble1, cdouble1, rozmiar / 8);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (double, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(adouble4, bdouble4, cdouble4, rozmiar / 2);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (double, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(adouble8, bdouble8, cdouble8, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (double, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	dodawanieCPU(adouble16, bdouble16, cdouble16, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas dodawania CPU (double, 16MiB) [ms]: %f\n\n", 1000.0 * (stopCPU - startCPU));

	startCPU = omp_get_wtime();
	mnozenieCPU(aint1, bint1, cint1, rozmiar / 4);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (int, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(aint4, bint4, cint4, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (int, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(aint8, bint8, cint8, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (int, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(aint16, bint16, cint16, rozmiar * 4);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (int, 16MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(afloat1, bfloat1, cfloat1, rozmiar / 4);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (float, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(afloat4, bfloat4, cfloat4, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (float, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(afloat8, bfloat8, cfloat8, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (float, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(afloat16, bfloat16, cfloat16, rozmiar * 4);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (float, 16MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(adouble1, bdouble1, cdouble1, rozmiar / 8);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (double, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(adouble4, bdouble4, cdouble4, rozmiar / 2);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (double, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(adouble8, bdouble8, cdouble8, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (double, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozenieCPU(adouble16, bdouble16, cdouble16, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia CPU (double, 16MiB) [ms]: %f\n\n", 1000.0 * (stopCPU - startCPU));

	startCPU = omp_get_wtime();
	potegowanieCPU(aint1, bint1, cint1, rozmiar / 4);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (int, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(aint4, bint4, cint4, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (int, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(aint8, bint8, cint8, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (int, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(aint16, bint16, cint16, rozmiar * 4);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (int, 16MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(afloat1, bfloat1, cfloat1, rozmiar / 4);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (float, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(afloat4, bfloat4, cfloat4, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (float, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(afloat8, bfloat8, cfloat8, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (float, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(afloat16, bfloat16, cfloat16, rozmiar * 4);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (float, 16MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(adouble1, bdouble1, cdouble1, rozmiar / 8);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (double, 1MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(adouble4, bdouble4, cdouble4, rozmiar / 2);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (double, 4MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(adouble8, bdouble8, cdouble8, rozmiar);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (double, 8MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	potegowanieCPU(adouble16, bdouble16, cdouble16, rozmiar * 2);
	stopCPU = omp_get_wtime();
	printf("Czas potegowania CPU (double, 16MiB) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));

	delete[] aint1;
	delete[] aint4;
	delete[] aint8;
	delete[] aint16;
	delete[] bint1;
	delete[] bint4;
	delete[] bint8;
	delete[] bint16;
	delete[] cint1;
	delete[] cint4;
	delete[] cint8;
	delete[] cint16;
	delete[] afloat1;
	delete[] afloat4;
	delete[] afloat8;
	delete[] afloat16;
	delete[] bfloat1;
	delete[] bfloat4;
	delete[] bfloat8;
	delete[] bfloat16;
	delete[] cfloat1;
	delete[] cfloat4;
	delete[] cfloat8;
	delete[] cfloat16;
	delete[] adouble1;
	delete[] adouble4;
	delete[] adouble8;
	delete[] adouble16;
	delete[] bdouble1;
	delete[] bdouble4;
	delete[] bdouble8;
	delete[] bdouble16;
	delete[] cdouble1;
	delete[] cdouble4;
	delete[] cdouble8;
	delete[] cdouble16;

	hipFree(dev_aint1);
	hipFree(dev_aint4);
	hipFree(dev_aint8);
	hipFree(dev_aint16);
	hipFree(dev_bint1);
	hipFree(dev_bint4);
	hipFree(dev_bint8);
	hipFree(dev_bint16);
	hipFree(dev_cint1);
	hipFree(dev_cint4);
	hipFree(dev_cint8);
	hipFree(dev_cint16);
	hipFree(dev_afloat1);
	hipFree(dev_afloat4);
	hipFree(dev_afloat8);
	hipFree(dev_afloat16);
	hipFree(dev_bfloat1);
	hipFree(dev_bfloat4);
	hipFree(dev_bfloat8);
	hipFree(dev_bfloat16);
	hipFree(dev_cfloat1);
	hipFree(dev_cfloat4);
	hipFree(dev_cfloat8);
	hipFree(dev_cfloat16);
	hipFree(dev_adouble1);
	hipFree(dev_adouble4);
	hipFree(dev_adouble8);
	hipFree(dev_adouble16);
	hipFree(dev_bdouble1);
	hipFree(dev_bdouble4);
	hipFree(dev_bdouble8);
	hipFree(dev_bdouble16);
	hipFree(dev_cdouble1);
	hipFree(dev_cdouble4);
	hipFree(dev_cdouble8);
	hipFree(dev_cdouble16);

	hipDeviceReset();
	return 0;
}

void dodawanieCPU(int *a, int *b, int *c, int rozmiar)
{
	for (int i = 0; i < rozmiar; i++)
	{
		c[i] = a[i] + b[i];
	}
}

void mnozenieCPU(int *a, int *b, int *c, int rozmiar)
{
	for (int i = 0; i < rozmiar; i++)
	{
		c[i] = a[i] * b[i];
	}
}

void potegowanieCPU(int *a, int *b, int *c, int rozmiar)
{
	int wynik;
	for (int i = 0; i < rozmiar; i++)
	{
		wynik = 1;
		for (int j = 0; j < b[i]; j++)
		{
			wynik *= a[i];
		}
		c[i] = wynik;
	}
}

void dodawanieCPU(float *a, float *b, float *c, int rozmiar)
{
	for (int i = 0; i < rozmiar; i++)
	{
		c[i] = a[i] + b[i];
}
}

void mnozenieCPU(float *a, float *b, float *c, int rozmiar)
{
	for (int i = 0; i < rozmiar; i++)
	{
		c[i] = a[i] * b[i];
	}
}

void potegowanieCPU(float *a, float *b, float *c, int rozmiar)
{
	float wynik;
	for (int i = 0; i < rozmiar; i++)
	{
		wynik = 1;
		for (float j = 0; j < b[i]; j++)
		{
			wynik *= a[i];
		}
		c[i] = wynik;
	}
}

void dodawanieCPU(double *a, double *b, double *c, int rozmiar)
{
	for (int i = 0; i < rozmiar; i++)
	{
		c[i] = a[i] + b[i];
	}
}

void mnozenieCPU(double *a, double *b, double *c, int rozmiar)
{
	for (int i = 0; i < rozmiar; i++)
	{
		c[i] = a[i] * b[i];
	}
}

void potegowanieCPU(double *a, double *b, double *c, int rozmiar)
{
	double wynik;
	for (int i = 0; i < rozmiar; i++)
	{
		wynik = 1;
		for (double j = 0; j < b[i]; j++)
		{
			wynik *= a[i];
		}
		c[i] = wynik;
	}
}


#endif

#ifdef Zad4
#include <math.h>
__global__ void kernelDodawanieMacierzy(float *a, float *b, float *c, int rozmiar)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < rozmiar && j < rozmiar)
	{
		c[i * rozmiar + j] = a[i * rozmiar + j] + b[i * rozmiar + j];
	}
}
__global__ void kernelMnozenieMacierzy(float *a, float *b, float *c, int rozmiar, int sqrtRozmiar)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	float wynik = 0;
	if (i < sqrtRozmiar && j < sqrtRozmiar)
	{
		/*for (int k = 0; k < rozmiar; k++)
		{
			wynik += a[i * rozmiar + k] * b[k * rozmiar + j];
		}*/
		c[i * rozmiar + j] = wynik;
	}
}
__global__ void kernelDodawanieMacierzy(double *a, double *b, double *c, int rozmiar)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < rozmiar && j < rozmiar)
	{
		c[i * rozmiar + j] = a[i * rozmiar + j] + b[i * rozmiar + j];
	}
}
__global__ void kernelMnozenieMacierzy(double *a, double *b, double *c, int rozmiar, int sqrtRozmiar)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	double wynik = 0;
	if (i < sqrtRozmiar && j < sqrtRozmiar)
	{
		for (int k = 0; k < rozmiar; k++)
		{
			wynik += a[i * rozmiar + k] * b[k * rozmiar + j];
		}
		c[i * rozmiar + j] = wynik;
	}
}

void dodawaniemacierzyCPU(float *a, float *b, float *c, int rozmiar);
void mnozeniemacierzyCPU(float *a, float *b, float *c, int rozmiar);
void dodawaniemacierzyCPU(double *a, double *b, double *c, int rozmiar);
void mnozeniemacierzyCPU(double *a, double *b, double *c, int rozmiar);

int main()
{
	hipSetDevice(0);
	double startCPU, stopCPU;
	const int rozmiar = 1024;
	int liczbaBlokow;
	float czasGPU;
	hipEvent_t startGPU, stopGPU;
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);

	float *afloat1 = new float[rozmiar * rozmiar / 4];
	float *dev_afloat1;
	hipMalloc(&dev_afloat1, rozmiar * rozmiar * sizeof(float) / 4);
	hipMemcpy(dev_afloat1, afloat1, rozmiar * rozmiar * sizeof(float) / 4, hipMemcpyHostToDevice);
	float *bfloat1 = new float[rozmiar * rozmiar / 4];
	float *dev_bfloat1;
	hipMalloc(&dev_bfloat1, rozmiar * rozmiar * sizeof(float) / 4);
	hipMemcpy(dev_bfloat1, bfloat1, rozmiar * rozmiar * sizeof(float) / 4, hipMemcpyHostToDevice);
	float *cfloat1 = new float[rozmiar * rozmiar / 4];
	float *dev_cfloat1;
	hipMalloc(&dev_cfloat1, rozmiar * rozmiar * sizeof(float) / 4);
	hipMemcpy(dev_cfloat1, cfloat1, rozmiar * rozmiar * sizeof(float) / 4, hipMemcpyHostToDevice);
	liczbaBlokow = (rozmiar * rozmiar / 4 + rozmiar - 1) / rozmiar;
	hipEventRecord(startGPU, 0);
	kernelDodawanieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_afloat1, dev_bfloat1, dev_cfloat1, rozmiar / 4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas dodawania macierzy GPU (float, 1) [ms]: %f\n", czasGPU);
	hipEventRecord(startGPU, 0);
	kernelMnozenieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_afloat1, dev_bfloat1, dev_cfloat1, rozmiar / 4, floor(sqrt(rozmiar / 4)) - 1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas mnozenia macierzy GPU (float, 1) [ms]: %f\n", czasGPU);
	startCPU = omp_get_wtime();
	dodawaniemacierzyCPU(afloat1, bfloat1, cfloat1, ceil(rozmiar * rozmiar / 4));
	stopCPU = omp_get_wtime();
	printf("Czas dodawania macierzy CPU (float, 1) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozeniemacierzyCPU(afloat1, bfloat1, cfloat1, ceil(rozmiar * rozmiar / 4));
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia macierzy CPU (float, 1) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	delete[] afloat1;
	hipFree(dev_afloat1);
	delete[] bfloat1;
	hipFree(dev_bfloat1);
	delete[] cfloat1;
	hipFree(dev_cfloat1);

	float *afloat4 = new float[rozmiar * rozmiar];
	float *dev_afloat4;
	hipMalloc(&dev_afloat4, rozmiar * rozmiar * sizeof(float));
	hipMemcpy(dev_afloat4, afloat4, rozmiar * rozmiar * sizeof(float), hipMemcpyHostToDevice);
	float *bfloat4 = new float[rozmiar * rozmiar];
	float *dev_bfloat4;
	hipMalloc(&dev_bfloat4, rozmiar * rozmiar * sizeof(float));
	hipMemcpy(dev_bfloat4, bfloat4, rozmiar * rozmiar * sizeof(float), hipMemcpyHostToDevice);
	float *cfloat4 = new float[rozmiar * rozmiar];
	float *dev_cfloat4;
	hipMalloc(&dev_cfloat4, rozmiar * rozmiar * sizeof(float));
	hipMemcpy(dev_cfloat4, cfloat4, rozmiar * rozmiar * sizeof(float), hipMemcpyHostToDevice);
	liczbaBlokow = (rozmiar * rozmiar + rozmiar - 1) / rozmiar;
	hipEventRecord(startGPU, 0);
	kernelDodawanieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_afloat4, dev_bfloat4, dev_cfloat4, rozmiar);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas dodawania macierzy GPU (float, 4) [ms]: %f\n", czasGPU);
	hipEventRecord(startGPU, 0);
	kernelMnozenieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_afloat4, dev_bfloat4, dev_cfloat4, rozmiar, floor(sqrt(rozmiar)) - 1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas mnozenia macierzy GPU (float, 4) [ms]: %f\n", czasGPU);
	startCPU = omp_get_wtime();
	dodawaniemacierzyCPU(afloat4, bfloat4, cfloat4, ceil(rozmiar * rozmiar));
	stopCPU = omp_get_wtime();
	printf("Czas dodawania macierzy CPU (float, 4) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozeniemacierzyCPU(afloat4, bfloat4, cfloat4, ceil(rozmiar * rozmiar));
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia macierzy CPU (float, 4) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	delete[] afloat4;
	hipFree(dev_afloat4);
	delete[] bfloat4;
	hipFree(dev_bfloat4);
	delete[] cfloat4;
	hipFree(dev_cfloat4);

	float *afloat8 = new float[rozmiar * rozmiar * 2];
	float *dev_afloat8;
	hipMalloc(&dev_afloat8, rozmiar * rozmiar * sizeof(float) * 2);
	hipMemcpy(dev_afloat8, afloat8, rozmiar * rozmiar * sizeof(float) * 2, hipMemcpyHostToDevice);
	float *bfloat8 = new float[rozmiar * rozmiar * 2];
	float *dev_bfloat8;
	hipMalloc(&dev_bfloat8, rozmiar * rozmiar * sizeof(float) * 2);
	hipMemcpy(dev_bfloat8, bfloat8, rozmiar * rozmiar * sizeof(float) * 2, hipMemcpyHostToDevice);
	float *cfloat8 = new float[rozmiar * rozmiar * 2];
	float *dev_cfloat8;
	hipMalloc(&dev_cfloat8, rozmiar * rozmiar * sizeof(float) * 2);
	hipMemcpy(dev_cfloat8, cfloat8, rozmiar * rozmiar * sizeof(float) * 2, hipMemcpyHostToDevice);
	liczbaBlokow = (rozmiar * rozmiar * 2 + rozmiar - 1) / rozmiar;
	hipEventRecord(startGPU, 0);
	kernelDodawanieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_afloat8, dev_bfloat8, dev_cfloat8, rozmiar * 2);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas dodawania macierzy GPU (float, 8) [ms]: %f\n", czasGPU);
	hipEventRecord(startGPU, 0);
	kernelMnozenieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_afloat8, dev_bfloat8, dev_cfloat8, rozmiar * 2, floor(sqrt(rozmiar * 2)) - 1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas mnozenia macierzy GPU (float, 8) [ms]: %f\n", czasGPU);
	startCPU = omp_get_wtime();
	dodawaniemacierzyCPU(afloat8, bfloat8, cfloat8, ceil(rozmiar * rozmiar * 2));
	stopCPU = omp_get_wtime();
	printf("Czas dodawania macierzy CPU (float, 8) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozeniemacierzyCPU(afloat8, bfloat8, cfloat8, ceil(rozmiar * rozmiar * 2));
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia macierzy CPU (float, 8) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	delete[] afloat8;
	hipFree(dev_afloat8);
	delete[] bfloat8;
	hipFree(dev_bfloat8);
	delete[] cfloat8;
	hipFree(dev_cfloat8);

	float *afloat16 = new float[rozmiar * rozmiar * 4];
	float *dev_afloat16;
	hipMalloc(&dev_afloat16, rozmiar * rozmiar * sizeof(float) * 4);
	hipMemcpy(dev_afloat16, afloat16, rozmiar * rozmiar * sizeof(float) * 4, hipMemcpyHostToDevice);
	float *bfloat16 = new float[rozmiar * rozmiar * 4];
	float *dev_bfloat16;
	hipMalloc(&dev_bfloat16, rozmiar * rozmiar * sizeof(float) * 4);
	hipMemcpy(dev_bfloat16, bfloat16, rozmiar * rozmiar * sizeof(float) * 4, hipMemcpyHostToDevice);
	float *cfloat16 = new float[rozmiar * rozmiar * 4];
	float *dev_cfloat16;
	hipMalloc(&dev_cfloat16, rozmiar * rozmiar * sizeof(float) * 4);
	hipMemcpy(dev_cfloat16, cfloat16, rozmiar * rozmiar * sizeof(float) * 4, hipMemcpyHostToDevice);
	liczbaBlokow = (rozmiar * rozmiar * 4 + rozmiar - 1) / rozmiar;
	hipEventRecord(startGPU, 0);
	kernelDodawanieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_afloat16, dev_bfloat16, dev_cfloat16, rozmiar * 4);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas dodawania macierzy GPU (float, 16) [ms]: %f\n", czasGPU);
	hipEventRecord(startGPU, 0);
	kernelMnozenieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_afloat16, dev_bfloat16, dev_cfloat16, rozmiar * 4, floor(sqrt(rozmiar * 4)) - 1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas mnozenia macierzy GPU (float, 16) [ms]: %f\n", czasGPU);
	startCPU = omp_get_wtime();
	dodawaniemacierzyCPU(afloat16, bfloat16, cfloat16, ceil(rozmiar * rozmiar * 4));
	stopCPU = omp_get_wtime();
	printf("Czas dodawania macierzy CPU (float, 16) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozeniemacierzyCPU(afloat16, bfloat16, cfloat16, ceil(rozmiar * rozmiar * 4));
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia macierzy CPU (float, 16) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	delete[] afloat16;
	hipFree(dev_afloat16);
	delete[] bfloat16;
	hipFree(dev_bfloat16);
	delete[] cfloat16;
	hipFree(dev_cfloat16);

	double *adouble1 = new double[rozmiar * rozmiar / 8];
	double *dev_adouble1;
	hipMalloc(&dev_adouble1, rozmiar * rozmiar * sizeof(double) / 8);
	hipMemcpy(dev_adouble1, adouble1, rozmiar * rozmiar * sizeof(double) / 8, hipMemcpyHostToDevice);
	double *bdouble1 = new double[rozmiar * rozmiar / 8];
	double *dev_bdouble1;
	hipMalloc(&dev_bdouble1, rozmiar * rozmiar * sizeof(double) / 8);
	hipMemcpy(dev_bdouble1, bdouble1, rozmiar * rozmiar * sizeof(double) / 8, hipMemcpyHostToDevice);
	double *cdouble1 = new double[rozmiar * rozmiar / 8];
	double *dev_cdouble1;
	hipMalloc(&dev_cdouble1, rozmiar * rozmiar * sizeof(double) / 8);
	hipMemcpy(dev_cdouble1, cdouble1, rozmiar * rozmiar * sizeof(double) / 8, hipMemcpyHostToDevice);
	liczbaBlokow = (rozmiar * rozmiar / 8 + rozmiar - 1) / rozmiar;
	hipEventRecord(startGPU, 0);
	kernelDodawanieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_adouble1, dev_bdouble1, dev_cdouble1, rozmiar / 8);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas dodawania macierzy GPU (double, 1) [ms]: %f\n", czasGPU);
	hipEventRecord(startGPU, 0);
	kernelMnozenieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_adouble1, dev_bdouble1, dev_cdouble1, rozmiar / 8, floor(sqrt(rozmiar / 8)) - 1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas mnozenia macierzy GPU (double, 1) [ms]: %f\n", czasGPU);
	startCPU = omp_get_wtime();
	dodawaniemacierzyCPU(adouble1, bdouble1, cdouble1, ceil(rozmiar * rozmiar / 8));
	stopCPU = omp_get_wtime();
	printf("Czas dodawania macierzy CPU (double, 1) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozeniemacierzyCPU(adouble1, bdouble1, cdouble1, ceil(rozmiar * rozmiar / 8));
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia macierzy CPU (double, 1) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	delete[] adouble1;
	hipFree(dev_adouble1);
	delete[] bdouble1;
	hipFree(dev_bdouble1);
	delete[] cdouble1;
	hipFree(dev_cdouble1);

	double *adouble4 = new double[rozmiar * rozmiar / 2];
	double *dev_adouble4;
	hipMalloc(&dev_adouble4, rozmiar * rozmiar * sizeof(double) / 2);
	hipMemcpy(dev_adouble4, adouble4, rozmiar * rozmiar * sizeof(double) / 2, hipMemcpyHostToDevice);
	double *bdouble4 = new double[rozmiar * rozmiar / 2];
	double *dev_bdouble4;
	hipMalloc(&dev_bdouble4, rozmiar * rozmiar * sizeof(double) / 2);
	hipMemcpy(dev_bdouble4, bdouble4, rozmiar * rozmiar * sizeof(double) / 2, hipMemcpyHostToDevice);
	double *cdouble4 = new double[rozmiar * rozmiar / 2];
	double *dev_cdouble4;
	hipMalloc(&dev_cdouble4, rozmiar * rozmiar * sizeof(double) / 2);
	hipMemcpy(dev_cdouble4, cdouble4, rozmiar * rozmiar * sizeof(double) / 2, hipMemcpyHostToDevice);
	liczbaBlokow = (rozmiar * rozmiar / 2 + rozmiar - 1) / rozmiar;
	hipEventRecord(startGPU, 0);
	kernelDodawanieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_adouble4, dev_bdouble4, dev_cdouble4, rozmiar / 2);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas dodawania macierzy GPU (double, 4) [ms]: %f\n", czasGPU);
	hipEventRecord(startGPU, 0);
	kernelMnozenieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_adouble4, dev_bdouble4, dev_cdouble4, rozmiar / 2, floor(sqrt(rozmiar / 2)) - 1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas mnozenia macierzy GPU (double, 4) [ms]: %f\n", czasGPU);
	startCPU = omp_get_wtime();
	dodawaniemacierzyCPU(adouble4, bdouble4, cdouble4, ceil(rozmiar * rozmiar / 2));
	stopCPU = omp_get_wtime();
	printf("Czas dodawania macierzy CPU (double, 4) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozeniemacierzyCPU(adouble4, bdouble4, cdouble4, ceil(rozmiar * rozmiar / 2));
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia macierzy CPU (double, 4) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	delete[] adouble4;
	hipFree(dev_adouble4);
	delete[] bdouble4;
	hipFree(dev_bdouble4);
	delete[] cdouble4;
	hipFree(dev_cdouble4);

	double *adouble8 = new double[rozmiar * rozmiar];
	double *dev_adouble8;
	hipMalloc(&dev_adouble8, rozmiar * rozmiar * sizeof(double));
	hipMemcpy(dev_adouble8, adouble8, rozmiar * rozmiar * sizeof(double), hipMemcpyHostToDevice);
	double *bdouble8 = new double[rozmiar * rozmiar];
	double *dev_bdouble8;
	hipMalloc(&dev_bdouble8, rozmiar * rozmiar * sizeof(double));
	hipMemcpy(dev_bdouble8, bdouble8, rozmiar * rozmiar * sizeof(double), hipMemcpyHostToDevice);
	double *cdouble8 = new double[rozmiar * rozmiar];
	double *dev_cdouble8;
	hipMalloc(&dev_cdouble8, rozmiar * rozmiar * sizeof(double));
	hipMemcpy(dev_cdouble8, cdouble8, rozmiar * rozmiar * sizeof(double), hipMemcpyHostToDevice);
	liczbaBlokow = (rozmiar * rozmiar + rozmiar - 1) / rozmiar;
	hipEventRecord(startGPU, 0);
	kernelDodawanieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_adouble8, dev_bdouble8, dev_cdouble8, rozmiar);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas dodawania macierzy GPU (double, 8) [ms]: %f\n", czasGPU);
	hipEventRecord(startGPU, 0);
	kernelMnozenieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_adouble8, dev_bdouble8, dev_cdouble8, rozmiar, floor(sqrt(rozmiar)) - 1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas mnozenia macierzy GPU (double, 8) [ms]: %f\n", czasGPU);
	startCPU = omp_get_wtime();
	dodawaniemacierzyCPU(adouble8, bdouble8, cdouble8, ceil(rozmiar * rozmiar));
	stopCPU = omp_get_wtime();
	printf("Czas dodawania macierzy CPU (double, 8) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozeniemacierzyCPU(adouble8, bdouble8, cdouble8, ceil(rozmiar * rozmiar));
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia macierzy CPU (double, 8) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	delete[] adouble8;
	hipFree(dev_adouble8);
	delete[] bdouble8;
	hipFree(dev_bdouble8);
	delete[] cdouble8;
	hipFree(dev_cdouble8);

	double *adouble16 = new double[rozmiar * rozmiar * 2];
	double *dev_adouble16;
	hipMalloc(&dev_adouble16, rozmiar * rozmiar * sizeof(double) * 2);
	hipMemcpy(dev_adouble16, adouble16, rozmiar * rozmiar * sizeof(double) * 2, hipMemcpyHostToDevice);
	double *bdouble16 = new double[rozmiar * rozmiar * 2];
	double *dev_bdouble16;
	hipMalloc(&dev_bdouble16, rozmiar * rozmiar * sizeof(double) * 2);
	hipMemcpy(dev_bdouble16, bdouble16, rozmiar * rozmiar * sizeof(double) * 2, hipMemcpyHostToDevice);
	double *cdouble16 = new double[rozmiar * rozmiar * 2];
	double *dev_cdouble16;
	hipMalloc(&dev_cdouble16, rozmiar * rozmiar * sizeof(double) * 2);
	hipMemcpy(dev_cdouble16, cdouble16, rozmiar * rozmiar * sizeof(double) * 2, hipMemcpyHostToDevice);
	liczbaBlokow = (rozmiar * rozmiar * 2 + rozmiar - 1) / rozmiar;
	hipEventRecord(startGPU, 0);
	kernelDodawanieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_adouble16, dev_bdouble16, dev_cdouble16, rozmiar * 2);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas dodawania macierzy GPU (double, 16) [ms]: %f\n", czasGPU);
	hipEventRecord(startGPU, 0);
	kernelMnozenieMacierzy << <dim3(liczbaBlokow, 1), dim3(rozmiar, 1) >> > (dev_adouble16, dev_bdouble16, dev_cdouble16, rozmiar * 2, floor(sqrt(rozmiar * 2)) - 1);
	hipEventRecord(stopGPU, 0);
	hipEventSynchronize(stopGPU);
	hipDeviceSynchronize();
	hipEventElapsedTime(&czasGPU, startGPU, stopGPU);
	printf("Czas mnozenia macierzy GPU (double, 16) [ms]: %f\n", czasGPU);
	startCPU = omp_get_wtime();
	dodawaniemacierzyCPU(adouble16, bdouble16, cdouble16, ceil(rozmiar * rozmiar * 2));
	stopCPU = omp_get_wtime();
	printf("Czas dodawania macierzy CPU (double, 16) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	startCPU = omp_get_wtime();
	mnozeniemacierzyCPU(adouble16, bdouble16, cdouble16, ceil(rozmiar * rozmiar * 2));
	stopCPU = omp_get_wtime();
	printf("Czas mnozenia macierzy CPU (double, 16) [ms]: %f\n", 1000.0 * (stopCPU - startCPU));
	delete[] adouble16;
	hipFree(dev_adouble16);
	delete[] bdouble16;
	hipFree(dev_bdouble16);
	delete[] cdouble16;
	hipFree(dev_cdouble16);

	hipDeviceReset();
	return 0;
}

void dodawaniemacierzyCPU(float *a, float *b, float *c, int rozmiar)
{
	int size = floor(sqrt(rozmiar)-1);
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			c[i * size + j] = a[i * size + j] + b[i * size + j];
		}
	}
}
void mnozeniemacierzyCPU(float *a, float *b, float *c, int rozmiar)
{
	int size = floor(sqrt(rozmiar)-1);
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			c[i * size + j] = 0;
			for (int k = 0; k < size; k++)
			{
				c[i * size + j] += a[i * size + k] * b[k * size + j];
			}
		}
	}
}
void dodawaniemacierzyCPU(double *a, double *b, double *c, int rozmiar)
{
	int size = floor(sqrt(rozmiar)-1);
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			c[i * size + j] = a[i * size + j] + b[i * size + j];
		}
	}
}
void mnozeniemacierzyCPU(double *a, double *b, double *c, int rozmiar)
{
	int size = floor(sqrt(rozmiar)-1);
	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			c[i * size + j] = 0;
			for (int k = 0; k < size; k++)
			{
				c[i * size + j] += a[i * size + k] * b[k * size + j];
			}
		}
	}
}

#endif

#ifdef Zad5

#endif
